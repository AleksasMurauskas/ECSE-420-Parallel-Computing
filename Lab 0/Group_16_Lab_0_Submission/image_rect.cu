#include "hip/hip_runtime.h"
#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

/*

*/


__global__ void rectify(unsigned const char* input, unsigned char* output, int nBytes, int nThreads) {
  for (int i=threadIdx.x; i<nBytes; i+=nThreads) {
		if (input[i] < 127) {
			output[i] = 127;
		} else {
			output[i] = input[i];
		}
	}
}

int main(int argc, char *argv[]) {
  if (argc < 4) {
      printf("Run the following command:\n ./rectify <input png> <output png> <# threads>");
      //exit(0)
  }

  int nThreads = atoi(argv[3]); // number of threads
  if (nThreads<=0){
      nThreads= 1;
  }
  if (nThreads > 1024) {
      printf("Max number of threads if 1024.");
      nThreads = 1024;
  }

  char* input_filename = argv[1];
  char* output_filename = argv[2];
  unsigned char* image, *output_image;
  unsigned height, width;
  unsigned error;

  error = lodepng_decode32_file(&image, &width, &height, input_filename);
  if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

  unsigned int image_size = width * height * 4 * sizeof(unsigned char);
  
  unsigned char* cuda_input, *cuda_output;
  hipMalloc((void**) & cuda_input, image_size);
  hipMalloc((void**) & cuda_output, image_size);

  hipMemcpy(cuda_input, image, image_size, hipMemcpyHostToDevice);

  unsigned char * cuda_inputCpy = cuda_input,
    * cuda_outputCpy = cuda_output;
    
  int nBytes = width * height * 4;
  //while (pxLeft > nThreads) {
    rectify <<< 1, nThreads >>> (cuda_inputCpy, cuda_outputCpy, nBytes, nThreads);

		//cuda_inputCpy += nThreads;
		//cuda_outputCpy += nThreads;
		//pxLeft -= nThreads;
  //}
  //rectify<<<1, pxLeft>>>(cuda_inputCpy, cuda_outputCpy);
  hipDeviceSynchronize();

  output_image = (unsigned char*)malloc(image_size);
  hipMemcpy(output_image, cuda_output, image_size, hipMemcpyDeviceToHost);
  lodepng_encode32_file(output_filename, output_image, width, height);
  
  hipFree(cuda_input);
  hipFree(cuda_output);
  free(output_image);

  return 0;
}